#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2009 Ziyan Zhou <zhou@ziyan.info>
 * Visit http://ziyan.info/tag/ritcg2slam/ for more information
 *
 * This file is part of SLAM Visualization for Player.
 *
 * This is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This software is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this software.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "map.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define MAP_TRAVERSABLE -100
#define MAP_INTRAVERSABLE 100
#define MAP_UNKNOWN 0
#define MAP_UNEXPLORED 120

#define LASER_RANGE_MAX 8.0
#define LASER_RESOLUTION (M_PI / 360.0)

__device__ float __cuda_map_normalize(float a) {
	while(a < -M_PI) a += M_PI + M_PI;
	while(a >= M_PI) a -= M_PI + M_PI;
	return a;
}


#define D1 0.2
#define D2 0.25
#define D3 0.3

__device__ float __cuda_map_laser_model(float range, float a, float d) {
	float g =  -0.005 * (range > LASER_RANGE_MAX ? LASER_RANGE_MAX : range) + 0.05;
	float n = 1.0 / (0.05 * 2.0 * M_PI) * exp(a*a/-2.0);
	float s = g * n;
	if ( d < range - D1 ) {
		return -s;
	} else if ( d < range + D1 ) {
		return -s + s / D1 * ( d - range + D1 );
	} else if ( d < range + D2 ) {
		return s;
	} else if ( d < range + D3 ) {
		return s - s / (D3 - D2) * ( d - range - D2 );
	} else {
		return 0.0;
	}
}

__global__ void __cuda_map_laser(char* map, int width, int height, float mpp, float x, float y, float bearing, float *ranges, int top, int left)
{	
	int w = blockIdx.x + left;
	int h = threadIdx.x + top;
	// convert to map coords
	int ww = w + width / 2;
	int hh = h + height / 2;
	if (ww < 0 || hh < 0 || ww >= width || hh >= height) return;
	
	
	float dw = mpp * w;
	float dh = mpp * h;
	float t = sqrt((x-dw)*(x-dw)+(y-dh)*(y-dh));
	if (t > LASER_RANGE_MAX) return;



	float da = __cuda_map_normalize(atan2(dh - y, dw - x));
	int i = (int)ceil(__cuda_map_normalize(da - bearing - LASER_RESOLUTION / 2.0) / LASER_RESOLUTION) + 180;
	if(i < 0 || i >= 361) return;

	float change = __cuda_map_laser_model(ranges[i], __cuda_map_normalize(da - (bearing + LASER_RESOLUTION * (i - 180))), t);
	if(change == 0) return;

	char &value = map[hh * width + ww];
	if(value == MAP_UNEXPLORED) value = MAP_UNKNOWN;
	
	float prob = (float)value / 100.0 + change;
	if(prob > 1.0) prob = 1.0;
	if(prob < -1.0) prob = -1.0;
	value = 100 * prob;
	if(value < MAP_TRAVERSABLE) value = MAP_TRAVERSABLE;
	if(value > MAP_INTRAVERSABLE) value  = MAP_INTRAVERSABLE;
}

float normalize(const float a) {
	float angle = a;
	while(angle < -M_PI) angle += M_PI + M_PI;
	while(angle >= M_PI) angle -= M_PI + M_PI;
	return angle;
}

void cuda_map_create(void **map, const int width, const int height)
{
	hipMalloc(map, sizeof(char) * width * height);
}

void cuda_map_free(void **map)
{
	hipFree(*map);
	*map = 0;
}

void cuda_map_sync_to_cpu(const void *src, char *dst, const int width, const int height)
{
	hipMemcpy(dst, src, sizeof(char) * width * height, hipMemcpyDeviceToHost);
}

void cuda_map_sync_to_gpu(void *dst, const char *src, const int width, const int height)
{
	hipMemcpy(dst, src, sizeof(char) * width * height, hipMemcpyHostToDevice);
}



void cuda_map_laser(void *map, const int width, const int height, const float mpp, const float x, const float y, const float yaw, const float ranges[361])
{
	float bearing = normalize(yaw);

	// calculate bounding box
	int left = (int)floor((x - LASER_RANGE_MAX) / mpp);
	int right = (int)ceil((x + LASER_RANGE_MAX) / mpp);
	int top = (int)floor((y - LASER_RANGE_MAX) / mpp);
	int bottom = (int)ceil((y + LASER_RANGE_MAX) / mpp);

	float *r = 0;
	hipMalloc(&r, sizeof(float) * 361);
	hipMemcpy(r, ranges, sizeof(float) * 361, hipMemcpyHostToDevice);

	__cuda_map_laser<<<right-left+1, bottom-top+1>>>((char*)map, width, height, mpp, x, y, bearing, r, top, left);
	hipFree(r);
}


